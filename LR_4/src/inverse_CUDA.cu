#include "hip/hip_runtime.h"
#include "../include/inverse_CUDA.cuh"
#include "../include/matrix_operation.h"

// ~~~ ПАРАЛЛЕЛЬНОЕ ВЫЧИСЛЕНИЕ (CUDA) ~~~

__global__ void CUDA::noZeroInverseMatrix(double* matrix, int size, int row) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < size && j < size && i == row)
        if (matrix[i * size + i] == 0)
            for (int k = i + 1; k < size; k++)
                if (matrix[k * size + i] != 0.0) {
                    matrix[i * size + j] += matrix[k * size + j];
                    return;
                }
}

__global__ void CUDA::normalizeInverseMatrix(double* matrix, double* identityMatrix, int size, int row) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < size && j < size) {
        if (i == row && i != j) {
            identityMatrix[i * size + j] /= matrix[i * size + i];
            matrix[i * size + j] /= matrix[i * size + i];
        }

        __syncthreads();

        if (i == row && i == j) {
            identityMatrix[i * size + j] /= matrix[i * size + i];
            matrix[i * size + j] /= matrix[i * size + i];
        }
    }
}

__global__ void CUDA::calculateInverseMatrix(double* matrix, double* identityMatrix, int size, int row) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < size && j < size)
        if (i != row) {
            identityMatrix[i * size + j] -= identityMatrix[row * size + j] * matrix[i * size + row];

            if (j != row)
                matrix[i * size + j] -= matrix[row * size + j] * matrix[i * size + row];
        }
}

// Нахождение обратной матрицы с использованием метода Гаусса-Жордана
bool CUDA::inverseMatrixParallelCUDA(double** matrix, int size, int threadsInBlock) {
    // Подготовка конфигурации CUDA
    dim3 numBlocks(threadsInBlock, threadsInBlock);
    dim3 blockSize((size + threadsInBlock - 1) / threadsInBlock, (size + threadsInBlock - 1) / threadsInBlock);

    // Выделение памяти для результирующей матрицы и её инициализация как единичной матрицы
    double **resultMatrix = MATRIX::memoryAlloc(size);
    MATRIX::generateMatrix(resultMatrix, size, MATRIX::IDENTITY);

    // Подготовка памяти на хосте
    int size_bytes = size * size * sizeof(double);
    auto* h_A = new double[size * size];
    auto* h_I = new double[size * size];

    // Копирование исходной и единичной матриц в одномерные массивы для параллельных вычислений
    MATRIX::converterMatrix(matrix, h_A, size, true);
    MATRIX::converterMatrix(resultMatrix, h_I, size, true);

    // Выделение памяти на устройстве
    double* d_A;
    double* d_I;
    hipMalloc((void**)&d_A, size_bytes);
    hipMalloc((void**)&d_I, size_bytes);

    // Копирование данных из хоста на устройство
    hipMemcpy(d_A, h_A, size_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_I, h_I, size_bytes, hipMemcpyHostToDevice);

    // Запуск параллельного алгоритма
    for (int row = 0; row < size; row++) {
        // Обработка строки, чтобы главный элемент был ненулевым
        CUDA::noZeroInverseMatrix<<<blockSize, numBlocks>>>(d_A, size, row);
        hipDeviceSynchronize();

        // Нормализация строки
        CUDA::normalizeInverseMatrix<<<blockSize, numBlocks>>>(d_A, d_I, size, row);
        hipDeviceSynchronize();

        // Вычитание текущей строки из остальных строк
        CUDA::calculateInverseMatrix<<<blockSize, numBlocks>>>(d_A, d_I, size, row);
        hipDeviceSynchronize();
    }

    // Копирование результатов обратно на хост
    hipMemcpy(h_A, d_A, size_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(h_I, d_I, size_bytes, hipMemcpyDeviceToHost);

    // Конвертирование результатов в двумерный массив
    MATRIX::converterMatrix(matrix, h_I, size, false);

    // Освобождение памяти, выделенной для результирующей матрицы
    MATRIX::memoryFree(resultMatrix, size);

    // Освобождение памяти на устройстве
    hipFree(d_A);
    hipFree(d_I);

    // Освобождение памяти на хосте
    delete[] h_A;
    delete[] h_I;

    return true;
}